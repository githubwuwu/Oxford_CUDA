#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <hip/hip_runtime.h>
//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

	x[tid] = (float)threadIdx.x;
}


//
// main code
//

int main(int argc, char **argv)
{
	float *h_x, *d_x;
	int   nblocks, nthreads, nsize, n;

	// ��ʼ��
	int  devID= findCudaDevice(argc, (const char**)argv);
	hipDeviceProp_t deviceProps;
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
	printf("CUDA device [%s]\n", deviceProps.name);

	// set number of blocks, and threads per block

	nblocks = 0;
	nthreads = 8;
	nsize = nblocks*nthreads;

	// allocate memory for array

	h_x = (float *)malloc(nsize*sizeof(float));
	checkCudaErrors(hipMalloc(&d_x, nsize*sizeof(float)));

	// execute kernel

	my_first_kernel << <nblocks, nthreads >> >(d_x);
	getLastCudaError("my_first_kernel execution failed\n");

	// copy back results and print them out

	checkCudaErrors(hipMemcpy(h_x, d_x, nsize*sizeof(float), hipMemcpyDeviceToHost));

	for (n = 0; n<nsize; n++) printf(" n,  x  =  %d  %f \n", n, h_x[n]);

	// free memory 

	checkCudaErrors(hipFree(d_x));
	free(h_x);

	// CUDA exit -- needed to flush printf write buffer
	//hipDeviceReset()��Ϊ��ˢ��printf??���֮ǰ����������������Դ��
	//����hipDeviceSynchronize()
	hipDeviceReset();

	return 0;

	//���ʹ��ͳһ�ڴ�Ļ� ֻ��Ҫ����һ���ڴ�
	//checkCudaErrors(hipMallocManaged(&x, nsize*sizeof(float)));
	// hipDeviceSynchronize();
}
